#include "hip/hip_runtime.h"
#include "fci_computer_gpu_kernels.cuh"
#include <hip/hip_runtime.h>
#include <iostream>

// Helper function for atomic add with double precision
__device__ double atomicAdd_double(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}


// Helper function for atomic add with hipDoubleComplex
__device__ void atomicAdd_complex(hipDoubleComplex* addr, hipDoubleComplex val) {
    atomicAdd_double(&(addr->x), val.x);
    atomicAdd_double(&(addr->y), val.y);
}


// __global__ void apply_individual_nbody1_accumulate_kernel(
//     const hipDoubleComplex coeff, 
//     const hipDoubleComplex* d_Cin, 
//     hipDoubleComplex* d_Cout, 
//     const int* d_sourcea,
//     const int* d_targeta,
//     const int* d_paritya,
//     const int* d_sourceb,
//     const int* d_targetb,
//     const int* d_parityb,
//     int nbeta_strs_,
//     int targeta_size,
//     int targetb_size,
//     int tensor_size) 
// {
//     int index1 = blockIdx.x * blockDim.x + threadIdx.x;
//     // int index2 = blockIdx.d_Cout * blockDim.d_Cout + threadIdx.d_Cout;
    
//     if (index1 < targeta_size) {

//         int ta_idx = d_targeta[index1] * nbeta_strs_;
//         int sa_idx = d_sourcea[index1] * nbeta_strs_;

//         hipDoubleComplex pref = hipCmul(coeff, make_hipDoubleComplex(d_paritya[index1], 0.0));

//         for (int j = 0; j < targetb_size; j++) {

//             hipDoubleComplex term = hipCmul(pref, make_hipDoubleComplex(d_parityb[j], 0.0));
//             term = hipCmul(term, d_Cin[sa_idx + d_sourceb[j]]);
//             // atomicAdd(&d_Cout[ta_idx + d_targetb[j]].x, term.x);
//             // atomicAdd(&d_Cout[ta_idx + d_targetb[j]].y, term.y);
//             d_Cout[ta_idx + d_targetb[j]].x += term.x;
//             // d_Cout[ta_idx + d_targetb[j]].y += term.y;

//         }


//     }
// }

// CUDA kernel
// __global__ void apply_individual_nbody1_accumulate_kernel(
//     const hipDoubleComplex coeff, 
//     const hipDoubleComplex* d_Cin, 
//     hipDoubleComplex* d_Cout, 
//     const int* d_sourcea,
//     const int* d_targeta,
//     const int* d_paritya,
//     const int* d_sourceb,
//     const int* d_targetb,
//     const int* d_parityb,
//     int nbeta_strs_,
//     int targeta_size,
//     int targetb_size,
//     int tensor_size) 
// {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;

//     if (idx < targeta_size) {
//         int ta_idx = d_targeta[idx] * nbeta_strs_;
//         int sa_idx = d_sourcea[idx] * nbeta_strs_;
//         hipDoubleComplex pref = hipCmul(coeff, make_hipDoubleComplex(d_paritya[idx], 0.0));

//         #pragma unroll
//         for (int j = 0; j < targetb_size; ++j) {
//             hipDoubleComplex term = hipCmul(pref, make_hipDoubleComplex(d_parityb[j], 0.0));
//             term = hipCmul(term, d_Cin[sa_idx + d_sourceb[j]]);
//             d_Cout[ta_idx + d_targetb[j]] = hipCadd(term,  d_Cout[ta_idx + d_targetb[j]]);
//         }
//     }
// }

// V2 about 2x faster
__global__ void apply_individual_nbody1_accumulate_kernel(
    const hipDoubleComplex coeff, 
    const hipDoubleComplex* d_Cin, 
    hipDoubleComplex* d_Cout, 
    const int* d_sourcea,
    const int* d_targeta,
    const hipDoubleComplex* d_paritya,
    const int* d_sourceb,
    const int* d_targetb,
    const hipDoubleComplex* d_parityb,
    int nbeta_strs_,
    int targeta_size,
    int targetb_size,
    int tensor_size) 
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < targeta_size) {
        int ta_idx = d_targeta[idx] * nbeta_strs_;
        int sa_idx = d_sourcea[idx] * nbeta_strs_;
        
        // hipDoubleComplex pref = hipCmul(coeff, make_hipDoubleComplex(d_paritya[idx], 0.0));
        hipDoubleComplex pref = hipCmul(coeff, d_paritya[idx]);

         if (idy < targetb_size)  {
            hipDoubleComplex term = hipCmul(pref, d_parityb[idy]);

            term = hipCmul(term, d_Cin[sa_idx + d_sourceb[idy]]);

            d_Cout[ta_idx + d_targetb[idy]] = hipCadd(term,  d_Cout[ta_idx + d_targetb[idy]]);
        }
    }
}


// V3 about same as V2
// __global__ void apply_individual_nbody1_accumulate_kernel(
//     const hipDoubleComplex coeff, 
//     const hipDoubleComplex* __restrict__ d_Cin, 
//     hipDoubleComplex* __restrict__ d_Cout, 
//     const int* __restrict__ d_sourcea,
//     const int* __restrict__ d_targeta,
//     const int* __restrict__ d_paritya,
//     const int* __restrict__ d_sourceb,
//     const int* __restrict__ d_targetb,
//     const int* __restrict__ d_parityb,
//     int nbeta_strs_,
//     int targeta_size,
//     int targetb_size,
//     int tensor_size) 
// {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     int idy = blockIdx.y * blockDim.y + threadIdx.y;

//     if (idx < targetb_size) {
//         int ta_idx = d_targeta[idx] * nbeta_strs_;
//         int sa_idx = d_sourcea[idx] * nbeta_strs_;

//         hipDoubleComplex pref = hipCmul(coeff, make_hipDoubleComplex(d_paritya[idx], 0.0));

//         if (idy < targetb_size)  {
//             hipDoubleComplex term = hipCmul(pref, make_hipDoubleComplex(d_parityb[idy], 0.0));
//             term = hipCmul(term, d_Cin[sa_idx + d_sourceb[idy]]);
            
//             d_Cout[ta_idx + d_targetb[idy]] = hipCadd(term,  d_Cout[ta_idx + d_targetb[idy]]);
//         }
//     }
// }

// __global__ void apply_individual_nbody1_accumulate_kernel(
//     const hipDoubleComplex coeff, 
//     const hipDoubleComplex* __restrict__ d_Cin, 
//     hipDoubleComplex* __restrict__ d_Cout, 
//     const int* __restrict__ d_sourcea,
//     const int* __restrict__ d_targeta,
//     const int* __restrict__ d_paritya,
//     const int* __restrict__ d_sourceb,
//     const int* __restrict__ d_targetb,
//     const int* __restrict__ d_parityb,
//     int nbeta_strs_,
//     int targeta_size,
//     int targetb_size,
//     int tensor_size) 
// {
//     // // Allocate shared memory for source, target, and parity arrays
//     // extern __shared__ int shared_mem[];
//     // int* shared_sourcea = shared_mem;
//     // int* shared_targeta = shared_mem + targeta_size;
//     // int* shared_paritya = shared_mem + 2 * targeta_size;
//     // int* shared_sourceb = shared_mem + 3 * targeta_size;
//     // int* shared_targetb = shared_mem + 3 * targeta_size + targetb_size;
//     // int* shared_parityb = shared_mem + 3 * targeta_size + 2 * targetb_size;

//     // int tid = threadIdx.x + threadIdx.y * blockDim.x;
    
//     // // Load data into shared memory
//     // for (int i = tid; i < targeta_size; i += blockDim.x * blockDim.y) {
//     //     shared_sourcea[i] = d_sourcea[i];
//     //     shared_targeta[i] = d_targeta[i];
//     //     shared_paritya[i] = d_paritya[i];
//     // }
//     // for (int i = tid; i < targetb_size; i += blockDim.x * blockDim.y) {
//     //     shared_sourceb[i] = d_sourceb[i];
//     //     shared_targetb[i] = d_targetb[i];
//     //     shared_parityb[i] = d_parityb[i];
//     // }

//     // // Synchronize to ensure all threads have loaded their data into shared memory
//     // __syncthreads();

//     // Grid-stride loop for processing elements
//     for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < targeta_size; idx += blockDim.x * gridDim.x) {
//         int ta_idx = d_targeta[idx] * nbeta_strs_;
//         int sa_idx = d_sourcea[idx] * nbeta_strs_;
        
//         hipDoubleComplex pref = hipCmul(coeff, make_hipDoubleComplex(d_paritya[idx], 0.0));

//         for (int idy = blockIdx.y * blockDim.y + threadIdx.y; idy < targetb_size; idy += blockDim.y * gridDim.y) {
//             hipDoubleComplex term = hipCmul(pref, make_hipDoubleComplex(d_parityb[idy], 0.0));
//             term = hipCmul(term, d_Cin[sa_idx + d_sourceb[idy]]);

//             d_Cout[ta_idx + d_targetb[idy]] = hipCadd(term, d_Cout[ta_idx + d_targetb[idy]]);
//         }
//     }
// }

// V2_atomic - thread-safe version using atomicAdd to prevent race conditions
__global__ void apply_individual_nbody1_accumulate_kernel_atomic(
    const hipDoubleComplex coeff, 
    const hipDoubleComplex* d_Cin, 
    hipDoubleComplex* d_Cout, 
    const int* d_sourcea,
    const int* d_targeta,
    const hipDoubleComplex* d_paritya,
    const int* d_sourceb,
    const int* d_targetb,
    const hipDoubleComplex* d_parityb,
    int nbeta_strs_,
    int targeta_size,
    int targetb_size,
    int tensor_size) 
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < targeta_size) {
        int ta_idx = d_targeta[idx] * nbeta_strs_;
        int sa_idx = d_sourcea[idx] * nbeta_strs_;
        
        hipDoubleComplex pref = hipCmul(coeff, d_paritya[idx]);

        if (idy < targetb_size) {
            hipDoubleComplex term = hipCmul(pref, d_parityb[idy]);
            term = hipCmul(term, d_Cin[sa_idx + d_sourceb[idy]]);

            // Thread-safe atomic accumulation
            int output_idx = ta_idx + d_targetb[idy];
            atomicAdd_double(&d_Cout[output_idx].x, term.x);
            atomicAdd_double(&d_Cout[output_idx].y, term.y);
        }
    }
}

void apply_individual_nbody1_accumulate_wrapper(
    const hipDoubleComplex coeff, 
    const hipDoubleComplex* d_Cin, 
    hipDoubleComplex* d_Cout, 
    const int* d_sourcea,
    const int* d_targeta,
    const hipDoubleComplex* d_paritya,
    const int* d_sourceb,
    const int* d_targetb,
    const hipDoubleComplex* d_parityb,
    int nbeta_strs_,
    int targeta_size,
    int targetb_size,
    int tensor_size) 
{
    // 2D grid configuration for the atomic kernel
    dim3 blockSize(16, 16);  // 16x16 = 256 threads per block
    dim3 gridSize((targeta_size + blockSize.x - 1) / blockSize.x,
                  (targetb_size + blockSize.y - 1) / blockSize.y);
    
    apply_individual_nbody1_accumulate_kernel_atomic<<<gridSize, blockSize>>>(
        coeff, d_Cin, d_Cout, d_sourcea, d_targeta, d_paritya, 
        d_sourceb, d_targetb, d_parityb, nbeta_strs_, 
        targeta_size, targetb_size, tensor_size);
   

    // Check for any errors launching the kernel
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Failed to launch apply_individual_nbody1_accumulate_kernel (error code " << hipGetErrorString(err) << ")!" << std::endl;
        throw std::runtime_error("Kernel launch failed");
    }

    // Wait for the kernel to complete and check for errors
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "Kernel execution failed (error code " << hipGetErrorString(err) << ")!" << std::endl;
        throw std::runtime_error("Kernel execution failed");
    }
}


__global__ void apply_individual_nbody1_accumulate_kernel_atomic_v2(
    const hipDoubleComplex coeff, 
    const hipDoubleComplex* __restrict__ d_Cin,        // NEW: __restrict__
    hipDoubleComplex* __restrict__ d_Cout,             // NEW: __restrict__
    const int* __restrict__ d_sourcea,                // NEW: __restrict__
    const int* __restrict__ d_targeta,
    const hipDoubleComplex* __restrict__ d_paritya,
    const int* __restrict__ d_sourceb,
    const int* __restrict__ d_targetb,
    const hipDoubleComplex* __restrict__ d_parityb,
    int nbeta_strs_,
    int targeta_size,
    int targetb_size,
    int tensor_size) 
{
    int total = targeta_size * targetb_size;           // NEW: Flatten 2D grid
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < total) {
        int idx = index / targetb_size;                // Recover original indices
        int idy = index % targetb_size;

        int ta_idx = d_targeta[idx] * nbeta_strs_;
        int sa_idx = d_sourcea[idx] * nbeta_strs_;

        hipDoubleComplex pref = hipCmul(coeff, d_paritya[idx]);
        hipDoubleComplex term = hipCmul(pref, d_parityb[idy]);
        term = hipCmul(term, d_Cin[sa_idx + d_sourceb[idy]]);

        int output_idx = ta_idx + d_targetb[idy];
        atomicAdd_double(&d_Cout[output_idx].x, term.x);
        atomicAdd_double(&d_Cout[output_idx].y, term.y);
    }
}

void apply_individual_nbody1_accumulate_wrapper_v2(
    const hipDoubleComplex coeff, 
    const hipDoubleComplex* d_Cin, 
    hipDoubleComplex* d_Cout, 
    const int* d_sourcea,
    const int* d_targeta,
    const hipDoubleComplex* d_paritya,
    const int* d_sourceb,
    const int* d_targetb,
    const hipDoubleComplex* d_parityb,
    int nbeta_strs_,
    int targeta_size,
    int targetb_size,
    int tensor_size) 
{
    int total = targeta_size * targetb_size;
    int blockSize = 256;
    int numBlocks = (total + blockSize - 1) / blockSize;
    
    apply_individual_nbody1_accumulate_kernel_atomic<<<numBlocks, blockSize>>>(
        coeff, d_Cin, d_Cout, d_sourcea, d_targeta, d_paritya, 
        d_sourceb, d_targetb, d_parityb, nbeta_strs_, 
        targeta_size, targetb_size, tensor_size);
    
    // Check for any errors launching the kernel
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Failed to launch apply_individual_nbody1_accumulate_kernel_atomic_v2 (error code " << hipGetErrorString(err) << ")!" << std::endl;
        throw std::runtime_error("Kernel launch failed");
    }

    // Wait for the kernel to complete and check for errors
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "Kernel execution failed (error code " << hipGetErrorString(err) << ")!" << std::endl;
        throw std::runtime_error("Kernel execution failed");
    }
}


/* This is only better if the colision rate is very high */

/*
__global__ void apply_individual_nbody1_accumulate_kernel_shared(
    const hipDoubleComplex coeff, 
    const hipDoubleComplex* __restrict__ d_Cin,
    hipDoubleComplex* __restrict__ d_Cout,
    const int* __restrict__ d_sourcea,
    const int* __restrict__ d_targeta,
    const hipDoubleComplex* __restrict__ d_paritya,
    const int* __restrict__ d_sourceb,
    const int* __restrict__ d_targetb,
    const hipDoubleComplex* __restrict__ d_parityb,
    int nbeta_strs_,
    int targeta_size,
    int targetb_size) 
{
    // Flatten the 2D grid to 1D, as in previous example
    int total = targeta_size * targetb_size;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int blockSize = blockDim.x;
    int tid = threadIdx.x;

    // Allocate shared memory for reduction:
    extern __shared__ int shared[]; // Dynamic shared mem: int + complex per thread
    int* s_idx = shared; // [blockDim.x] -- each thread's output_idx
    hipDoubleComplex* s_val = (hipDoubleComplex*)&s_idx[blockSize]; // [blockDim.x] -- each thread's value

    // Each thread computes its term and output_idx
    int output_idx = -1;
    hipDoubleComplex term = make_hipDoubleComplex(0.0, 0.0);

    if (index < total) {
        int idx = index / targetb_size;
        int idy = index % targetb_size;

        int ta_idx = d_targeta[idx] * nbeta_strs_;
        int sa_idx = d_sourcea[idx] * nbeta_strs_;

        hipDoubleComplex pref = hipCmul(coeff, d_paritya[idx]);
        term = hipCmul(pref, d_parityb[idy]);
        term = hipCmul(term, d_Cin[sa_idx + d_sourceb[idy]]);

        output_idx = ta_idx + d_targetb[idy];
    }

    // Store each thread's output index and value in shared memory
    s_idx[tid] = output_idx;
    s_val[tid] = term;

    __syncthreads();

    // **Block-wise reduction by output_idx**
    // Each thread checks if it's the first occurrence of its output_idx in this block
    // If so, it sums all contributions in the block with that output_idx
    if (output_idx >= 0) {
        hipDoubleComplex block_sum = s_val[tid];

        // Only the first occurrence of this output_idx in the block performs the atomicAdd
        bool is_first = true;
        for (int t = 0; t < tid; ++t) {
            if (s_idx[t] == output_idx) {
                is_first = false;
                break;
            }
        }
        if (is_first) {
            // Sum all other threads in the block with the same output_idx
            for (int t = tid + 1; t < blockSize; ++t) {
                if (s_idx[t] == output_idx) {
                    block_sum.x += s_val[t].x;
                    block_sum.y += s_val[t].y;
                }
            }
            // One atomic add per unique output_idx per block
            atomicAdd_complex(&d_Cout[output_idx], block_sum);
        }
    }
}


extern "C" void apply_individual_nbody1_accumulate_wrapper_shared(
    const hipDoubleComplex coeff, 
    const hipDoubleComplex* d_Cin, 
    hipDoubleComplex* d_Cout, 
    const int* d_sourcea,
    const int* d_targeta,
    const hipDoubleComplex* d_paritya,
    const int* d_sourceb,
    const int* d_targetb,
    const hipDoubleComplex* d_parityb,
    int nbeta_strs_,
    int targeta_size,
    int targetb_size,
    int tensor_size) 
{
    int blockSize = 256;
    int numBlocks = (targeta_size * targetb_size + blockSize - 1) / blockSize;

    // Allocate shared memory for reduction
    size_t sharedMemSize = blockSize * (sizeof(int) + sizeof(hipDoubleComplex));

    apply_individual_nbody1_accumulate_kernel_shared<<<numBlocks, blockSize, sharedMemSize>>>(
        coeff, d_Cin, d_Cout, d_sourcea, d_targeta, d_paritya, 
        d_sourceb, d_targetb, d_parityb, nbeta_strs_, 
        targeta_size, targetb_size);
    
    // Check for any errors launching the kernel
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Failed to launch apply_individual_nbody1_accumulate_kernel_shared (error code " << hipGetErrorString(err) << ")!" << std::endl;
        throw std::runtime_error("Kernel launch failed");
    }

    // Wait for the kernel to complete and check for errors
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "Kernel execution failed (error code " << hipGetErrorString(err) << ")!" << std::endl;
        throw std::runtime_error("Kernel execution failed");
    }
}
    */

__global__ void scale_elements_kernel(
    hipDoubleComplex* d_Cout,
    const int* d_first, 
    int first_size,
    const int* d_second, 
    int second_size,
    int nbeta_strs_,
    hipDoubleComplex factor) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < first_size && j < second_size) {
        int idx = d_first[i] * nbeta_strs_ + d_second[j];
        d_Cout[idx] = hipCmul(d_Cout[idx], factor);
    }
}

extern "C" void scale_elements_wrapper(
    hipDoubleComplex* d_Cout,
    const int* d_first, 
    int first_size,
    const int* d_second, 
    int second_size,
    int nbeta_strs_,
    hipDoubleComplex factor) 
{
    dim3 blockSize(16, 16);
    dim3 gridSize((first_size + blockSize.x - 1) / blockSize.x, 
                  (second_size + blockSize.y - 1) / blockSize.y);

    scale_elements_kernel<<<gridSize, blockSize>>>(d_Cout, d_first, first_size, d_second, second_size, nbeta_strs_, factor);

    // Check for any errors launching the kernel
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Failed to launch scale_elements_kernel (error code " << hipGetErrorString(err) << ")!" << std::endl;
        throw std::runtime_error("Kernel launch failed");
    }

    // Wait for the kernel to complete and check for errors
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "Kernel execution failed (error code " << hipGetErrorString(err) << ")!" << std::endl;
        throw std::runtime_error("Kernel execution failed");
    }
}