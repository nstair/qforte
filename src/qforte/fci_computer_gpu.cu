#include "hip/hip_runtime.h"
#include "fci_computer_gpu.cuh"
#include <hip/hip_runtime.h>
#include <iostream>


// __device__ double2 cuDoubleComplexToDouble2(hipDoubleComplex z) {
//     double2 d;
//     d.x = hipCreal(z);
//     d.y = hipCimag(z);
//     return d;
// }

// __device__ hipDoubleComplex double2ToCuDoubleComplex(double2 d) {
//     return make_hipDoubleComplex(d.x, d.y);
// }

// __global__ void apply_individual_nbody1_accumulate_kernel(
//     const hipDoubleComplex coeff, 
//     const hipDoubleComplex* d_Cin, 
//     hipDoubleComplex* d_Cout, 
//     const int* d_sourcea,
//     const int* d_targeta,
//     const int* d_paritya,
//     const int* d_sourceb,
//     const int* d_targetb,
//     const int* d_parityb,
//     int nbeta_strs_,
//     int targeta_size,
//     int targetb_size,
//     int tensor_size) 
// {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;

//     if (idx < targeta_size) {
//         int ta_idx = d_targeta[idx] * nbeta_strs_;
//         int sa_idx = d_sourcea[idx] * nbeta_strs_;

//         hipDoubleComplex pref = hipCmul(coeff, make_hipDoubleComplex(d_paritya[idx], 0.0));

//         for (int j = 0; j < targetb_size; ++j) {
//             hipDoubleComplex term = hipCmul(pref, make_hipDoubleComplex(d_parityb[j], 0.0));
//             term = hipCmul(term, d_Cin[sa_idx + d_sourceb[j]]);

//             double2 term_double2 = cuDoubleComplexToDouble2(term);
//             double2* d_Cout_double2 = reinterpret_cast<double2*>(&d_Cout[ta_idx + d_targetb[j]]);

//             atomicAdd(&d_Cout_double2->x, term_double2.x);
//             atomicAdd(&d_Cout_double2->y, term_double2.y);
//         }
//     }
// }


// CUDA kernel
__global__ void apply_individual_nbody1_accumulate_kernel(
    const hipDoubleComplex coeff, 
    const hipDoubleComplex* d_Cin, 
    hipDoubleComplex* d_Cout, 
    const int* d_sourcea,
    const int* d_targeta,
    const int* d_paritya,
    const int* d_sourceb,
    const int* d_targetb,
    const int* d_parityb,
    int nbeta_strs,
    int targeta_size,
    int targetb_size,
    int tensor_size) 
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < targeta_size) {

        int ta_idx = d_targeta[idx] * nbeta_strs;

        int sa_idx = d_sourcea[idx] * nbeta_strs;

        hipDoubleComplex pref = hipCmul(coeff, make_hipDoubleComplex(d_paritya[idx], 0.0));

        for (int j = 0; j < targetb_size; ++j) {

            hipDoubleComplex term = hipCmul(pref, make_hipDoubleComplex(d_parityb[j], 0.0));

            term = hipCmul(term, d_Cin[sa_idx + d_sourceb[j]]);

            // atomicAdd(&d_Cout[ta_idx + d_targetb[j]], term);
            d_Cout[ta_idx + d_targetb[j]] = hipCadd(term,  d_Cout[ta_idx + d_targetb[j]]);

        }
    }
}

void apply_individual_nbody1_accumulate_wrapper(
    const hipDoubleComplex coeff, 
    const hipDoubleComplex* d_Cin, 
    hipDoubleComplex* d_Cout, 
    const int* d_sourcea,
    const int* d_targeta,
    const int* d_paritya,
    const int* d_sourceb,
    const int* d_targetb,
    const int* d_parityb,
    int nbeta_strs,
    int targeta_size,
    int targetb_size,
    int tensor_size) 
{
    int blocksPerGrid = (tensor_size + 256 - 1) / 256;
    apply_individual_nbody1_accumulate_kernel<<<blocksPerGrid, 256>>>(
        coeff, 
        d_Cin, 
        d_Cout, 
        d_sourcea, 
        d_targeta, 
        d_paritya, 
        d_sourceb, 
        d_targetb, 
        d_parityb, 
        nbeta_strs, 
        targeta_size, 
        targetb_size, 
        tensor_size);
   

    // Check for any errors launching the kernel
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Failed to launch apply_individual_nbody1_accumulate_kernel (error code " << hipGetErrorString(err) << ")!" << std::endl;
        throw std::runtime_error("Kernel launch failed");
    }

    // Wait for the kernel to complete and check for errors
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "Kernel execution failed (error code " << hipGetErrorString(err) << ")!" << std::endl;
        throw std::runtime_error("Kernel execution failed");
    }
}
