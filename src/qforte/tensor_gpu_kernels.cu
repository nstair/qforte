#include "hip/hip_runtime.h"
#include "tensor_gpu_kernels.cuh"
#include <hip/hip_runtime.h>
#include <iostream>

// Thrust includes for device vector operations
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <thrust/functional.h>


// Kernel for adding two tensors
__global__ void add_kernel(hipDoubleComplex* x, const hipDoubleComplex* y, size_t n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        //x[index] = x[index] + y[index];
        x[index] = hipCadd(x[index], y[index]);
    }
}

__global__ void add_kernel2(hipDoubleComplex* x, const hipDoubleComplex* y, size_t n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        //x[index] = x[index] + y[index];
        x[index] = hipCadd(x[index], y[index]);

    }
}

void add_wrapper(std::complex<double>* x, const std::complex<double>* y, size_t n, int threadsPerBlock) {
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    hipDoubleComplex *d_x, *d_y;
    hipMalloc(&d_x, n * sizeof(hipDoubleComplex));
    hipMalloc(&d_y, n * sizeof(hipDoubleComplex));

    hipDoubleComplex *h_x = new hipDoubleComplex[n];
    hipDoubleComplex *h_y = new hipDoubleComplex[n];
    for (size_t i = 0; i < n; ++i) {
        h_x[i] = make_hipDoubleComplex(x[i].real(), x[i].imag());
        h_y[i] = make_hipDoubleComplex(y[i].real(), y[i].imag());
    }

    hipMemcpy(d_x, h_x, n * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, n * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

    add_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_x, d_y, n);

    hipMemcpy(h_x, d_x, n * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    for (size_t i = 0; i < n; ++i) {
        x[i] = std::complex<double>(hipCreal(h_x[i]), hipCimag(h_x[i]));
    }

    hipFree(d_x);
    hipFree(d_y);
    delete[] h_x;
    delete[] h_y;
}


void add_wrapper2(hipDoubleComplex* d_x, const hipDoubleComplex* d_y, int n, int threadsPerBlock) {
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    // hipDoubleComplex val1 = make_hipDoubleComplex(d_x.real(), d_x.imag());
    // hipDoubleComplex val2 = make_hipDoubleComplex(d_y.real(), d_y.imag());

    // add_kernel2<<<blocksPerGrid, threadsPerBlock>>>(d_x, d_y, n);

    add_kernel2<<<blocksPerGrid, threadsPerBlock>>>(d_x, d_y, n);
   
}


// Custom functor for complex addition
struct complex_add {
    __host__ __device__
    hipDoubleComplex operator()(const hipDoubleComplex& a, const hipDoubleComplex& b) {
        return make_hipDoubleComplex(
            hipCreal(a) + hipCreal(b),
            hipCimag(a) + hipCimag(b)
        );
    }
};

// Using Thrust's transform function with custom functor
void add_wrapper_thrust(hipDoubleComplex* d_x, const hipDoubleComplex* d_y, size_t n) {
    thrust::device_ptr<hipDoubleComplex> dev_x(d_x);
    thrust::device_ptr<const hipDoubleComplex> dev_y(d_y);

    // Use our custom functor instead of hipCadd (which is problematic as a functor)
    thrust::transform(dev_x, dev_x + n, dev_y, dev_x, complex_add());
}