#include <hip/hip_runtime.h>
#include <stdint.h>
#include <hip/hip_complex.h>
#include <cstdio>

__device__ inline uint64_t set_bit_gpu(uint64_t n, int idx) {
    return n | (1ULL << idx);
}

__device__ inline uint64_t unset_bit_gpu(uint64_t n, int idx) {
    return n & ~(1ULL << idx);
}

__device__ inline int count_bits_above_gpu(uint64_t n, int idx) {
    uint64_t mask = (1ULL << idx) - 1;
    return __popcll(n & ~mask);
}

// GPU device function to perform binary search for index mapping
__device__ int binary_search_index_map(const uint64_t* keys, const int* values, int size, uint64_t target) {
    int left = 0, right = size - 1;
    while (left <= right) {
        int mid = (left + right) / 2;
        if (keys[mid] == target) {
            return values[mid];
        } else if (keys[mid] < target) {
            left = mid + 1;
        } else {
            right = mid - 1;
        }
    }
    return -1; // Not found
}

__global__ void make_mapping_each_gpu_kernel(
    const uint64_t* strings,
    const uint64_t* map_keys,
    const int* map_values,
    int map_size,
    const int* dag,
    const int* undag,
    int dag_size,
    int undag_size,
    uint64_t dag_mask,
    uint64_t undag_mask,
    int length,
    int* source,
    int* target,
    hipDoubleComplex* parity,
    int* count)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= length) return;

    uint64_t current = strings[index];
    
    // Check if the operator can act on this determinant
    bool check = ((current & dag_mask) == 0) && ((current & undag_mask) == undag_mask);
    
    if (check) {
        uint64_t parity_value = 0;
        uint64_t result_string = current;
        
        // Apply annihilation operators (undag) - process in reverse order
        for (int i = undag_size - 1; i >= 0; i--) {
            parity_value += count_bits_above_gpu(result_string, undag[i]);
            result_string = unset_bit_gpu(result_string, undag[i]);
        }
        
        // Apply creation operators (dag) - process in reverse order
        for (int i = dag_size - 1; i >= 0; i--) {
            parity_value += count_bits_above_gpu(result_string, dag[i]);
            result_string = set_bit_gpu(result_string, dag[i]);
        }
        
        // Find the target index using binary search
        int target_index = binary_search_index_map(map_keys, map_values, map_size, result_string);
        
        if (target_index >= 0) {
            // Atomically increment count and get insertion position
            int pos = atomicAdd(count, 1);
            
            // Store results
            source[pos] = index;
            target[pos] = target_index;
            
            // Convert parity from 0/1 to +1/-1 format
            int parity_int = 1 - 2 * static_cast<int>(parity_value % 2);
            parity[pos].x = static_cast<double>(parity_int);
            parity[pos].y = 0.0;
        }
    }
}

extern "C" void make_mapping_each_kernel_wrapper(
    const uint64_t* d_strings,
    const uint64_t* d_map_keys,
    const int* d_map_values,
    int map_size,
    const int* d_dag,
    const int* d_undag,
    int dag_size,
    int undag_size,
    uint64_t dag_mask,
    uint64_t undag_mask,
    int length,
    int* d_source,
    int* d_target,
    hipDoubleComplex* d_parity,
    int* d_count)
{
    // Calculate grid and block sizes
    int block_size = 256;
    int grid_size = (length + block_size - 1) / block_size;
    
    // Launch kernel
    make_mapping_each_gpu_kernel<<<grid_size, block_size>>>(
        d_strings, d_map_keys, d_map_values, map_size,
        d_dag, d_undag, dag_size, undag_size,
        dag_mask, undag_mask, length,
        d_source, d_target, d_parity, d_count
    );
    
    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
    }
    
    // Synchronize to ensure kernel completion
    hipDeviceSynchronize();
}
