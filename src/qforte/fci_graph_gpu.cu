#include "hip/hip_runtime.h"
#include "fci_graph_gpu.cuh"
#include <hip/hip_runtime.h>
#include <iostream>


__device__ uint64_t set_bit(uint64_t mask, int pos) {
    return mask | (1ULL << pos);
}

__device__ uint64_t unset_bit(uint64_t mask, int pos) {
    return mask & ~(1ULL << pos);
}

__device__ int count_bits_above(uint64_t number, int pos) {
    return __popc(number & ~((1ULL << (pos + 1)) - 1));
}

__device__ bool contains(const int* array, int size, int value) {
    for (int i = 0; i < size; ++i) {
        if (array[i] == value) {
            return true;
        }
    }
    return false;
}

__global__ void make_mapping_each_kernel(
    const int* dag,
    const int dag_size,
    const int* undag,
    const int undag_size,
    const uint64_t* strings,
    int length,
    int* source,
    int* target,
    int* parity,
    int* count)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= length) return;

    uint64_t dag_mask = 0;
    uint64_t undag_mask = 0;

    // Build dag_mask
    for (int i = 0; i < dag_size; i++) {
        if (!contains(undag, undag_size, dag[i])) {
            dag_mask = set_bit(dag_mask, dag[i]);
        }
    }

    // Build undag_mask
    for (int i = 0; i < undag_size; i++) {
        undag_mask = set_bit(undag_mask, undag[i]);
    }

    uint64_t current = strings[index];
    bool check = ((current & dag_mask) == 0) && ((current & (undag_mask ^ undag_mask)) == 0);

    if (check) {
        uint64_t tmp = current;
        uint64_t parity_value = 0;

        // Compute parity_value for undag
        for (int i = undag_size - 1; i >= 0; i--) {
            parity_value += count_bits_above(current, undag[i]);
            current = unset_bit(current, undag[i]);
        }

        // Compute parity_value for dag
        for (int i = dag_size - 1; i >= 0; i--) {
            parity_value += count_bits_above(current, dag[i]);
            current = set_bit(current, dag[i]);
        }

        int local_index = atomicAdd(count, 1);
        source[local_index] = index;
        target[local_index] = static_cast<int>(current);
        parity[local_index] = static_cast<int>(parity_value % 2);
    }
}

void make_mapping_each_wrapper(
    const int* dag,
    const int dag_size,
    const int* undag,
    const int undag_size,
    const uint64_t* strings,
    int length,
    int* source,
    int* target,
    int* parity,
    int* count) 
{

    int blockSize = 256; 
    int numBlocks = (length + blockSize - 1) / blockSize;

    make_mapping_each_kernel<<<numBlocks, blockSize>>>(
        d_dag, 
        dag.size(),
        d_undag, 
        undag.size(),
        d_strings, 
        length,
        d_source, 
        d_target, 
        d_parity, 
        d_count);
}