#include "hip/hip_runtime.h"
#include "tensor_gpu_kernels.cuh"
#include <hip/hip_runtime.h>
#include <iostream>


// Kernel for adding two tensors
__global__ void add_kernel(hipDoubleComplex* x, const hipDoubleComplex* y, size_t n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        //x[index] = x[index] + y[index];
        x[index] = hipCadd(x[index], y[index]);
    }
}

__global__ void add_kernel2(hipDoubleComplex* x, const hipDoubleComplex* y, size_t n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        //x[index] = x[index] + y[index];
        x[index] = hipCadd(x[index], y[index]);

    }
}

void add_wrapper(std::complex<double>* x, const std::complex<double>* y, size_t n, int threadsPerBlock) {
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    hipDoubleComplex *d_x, *d_y;
    hipMalloc(&d_x, n * sizeof(hipDoubleComplex));
    hipMalloc(&d_y, n * sizeof(hipDoubleComplex));

    hipDoubleComplex *h_x = new hipDoubleComplex[n];
    hipDoubleComplex *h_y = new hipDoubleComplex[n];
    for (size_t i = 0; i < n; ++i) {
        h_x[i] = make_hipDoubleComplex(x[i].real(), x[i].imag());
        h_y[i] = make_hipDoubleComplex(y[i].real(), y[i].imag());
    }

    hipMemcpy(d_x, h_x, n * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, n * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

    add_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_x, d_y, n);

    hipMemcpy(h_x, d_x, n * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    for (size_t i = 0; i < n; ++i) {
        x[i] = std::complex<double>(hipCreal(h_x[i]), hipCimag(h_x[i]));
    }

    hipFree(d_x);
    hipFree(d_y);
    delete[] h_x;
    delete[] h_y;
}


void add_wrapper2(hipDoubleComplex* d_x, const hipDoubleComplex* d_y, int n, int threadsPerBlock) {
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    // hipDoubleComplex val1 = make_hipDoubleComplex(d_x.real(), d_x.imag());
    // hipDoubleComplex val2 = make_hipDoubleComplex(d_y.real(), d_y.imag());

    // add_kernel2<<<blocksPerGrid, threadsPerBlock>>>(d_x, d_y, n);

    add_kernel2<<<blocksPerGrid, threadsPerBlock>>>(d_x, d_y, n);
   
}

